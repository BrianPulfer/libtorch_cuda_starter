#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <torch/torch.h>
#include <iostream>

using namespace std;

__global__ void sumKernel(const float* src, float* dst, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){ 
        atomicAdd(dst, src[idx]);
    }
}

float sumInterface(const torch::Tensor& src){
    int n = src.numel();
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Creating float (0.0) on device
    float *result = new float(0.0f);
    float *dst;
    hipMalloc(&dst, sizeof(float));
    hipMemcpy(dst, result, sizeof(float), hipMemcpyHostToDevice);

    // Launching kernel
    sumKernel<<<numBlocks, blockSize>>>(src.data_ptr<float>(), dst, n);
    
    // Copying back to host
    hipMemcpy(result, dst, sizeof(float), hipMemcpyDeviceToHost);
    return *result;
}

int main(int argc, char* argv[]){
    torch::Tensor tensor = torch::randn({2, 3}).to(at::kCUDA);

    cout << "Kernel Sum: " << sumInterface(tensor) << endl;
    cout << "Torchlib Sum: " << tensor.sum().item<float>() << endl;
    return 0;
}