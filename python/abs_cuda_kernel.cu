#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include <vector>


__global__ void abs_cuda_forward_kernel(
    const float* input,
    float* output,
    const int n
) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    output[index] = fabs(input[index]);
  }
}

std::vector<at::Tensor> abs_cuda_forward(
    torch::Tensor input
) {
  auto output = torch::empty_like(input);

  const int n = input.numel();
  const int blockSize = 256;
  const int numBlocks = (n + blockSize - 1) / blockSize;

  abs_cuda_forward_kernel<<<numBlocks, blockSize>>>(
      input.data_ptr<float>(),
      output.data_ptr<float>(),
      n
  );

  return {output};
}